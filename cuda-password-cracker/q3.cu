
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define PASSWORD_LENGTH 4
#define ENCRYPTED_LENGTH 11
#define TOTAL_COMBINATIONS (26 * 26 * 10 * 10)



__device__ void crypt(char * rawPassword, char * newPassword){
	newPassword[0] = rawPassword[0] + 3;
	newPassword[1] = rawPassword[0] - 2;
	newPassword[2] = rawPassword[0] + 1;
	newPassword[3] = rawPassword[1] + 1;
	newPassword[4] = rawPassword[1] - 2;
	newPassword[5] = rawPassword[1] - 3;
	newPassword[6] = rawPassword[2] + 1;
	newPassword[7] = rawPassword[2] - 2;
	newPassword[8] = rawPassword[3] + 4;
	newPassword[9] = rawPassword[3] - 3;
	newPassword[10] = '\0';
	
	for(int i =0; i<10; i++){
		if(i >= 0 && i < 6){ //checking all lower case letter limits
			if(newPassword[i] > 122){
				newPassword[i] = (newPassword[i] - 122) + 97;
			}else if(newPassword[i] < 97){
				newPassword[i] = (97 - newPassword[i]) + 97;
			}
		}else{ //checking number section
			if(newPassword[i] > 57){
				newPassword[i] = (newPassword[i] - 57) + 48;
			}else if(newPassword[i] < 48){
				newPassword[i] = (48 - newPassword[i]) + 48;
			}
		}
	}
}

__global__ void kernel(char * password, char * result, int * found){
	int threadID = blockDim.x * blockIdx.x + threadIdx.x;
	if (threadID >= TOTAL_COMBINATIONS || *found) {
		return;
	}
	    char rawPassword[PASSWORD_LENGTH + 1];
	    rawPassword[0] = 'a' + (threadID / (26 * 10 * 10)) % 26;
	    rawPassword[1] = 'a' + (threadID / (10 * 10)) % 26;
	    rawPassword[2] = '0' + (threadID / 10) % 10;
	    rawPassword[3] = '0' + threadID % 10;
	    rawPassword[4] = '\0';
	
	char encryptedGuess[ENCRYPTED_LENGTH];
    	crypt(rawPassword, encryptedGuess);

	    int match = 1;
	    for (int i = 0; i < ENCRYPTED_LENGTH; i++) {
		if (encryptedGuess[i] != password[i]) {
		    match = 0;
		    break;
		}
	    }

	    if (match == 1) {
		*found = 1;
		for (int i = 0; i < PASSWORD_LENGTH; i++) {
		    result[i] = rawPassword[i];
		}
		result[PASSWORD_LENGTH] = '\0';
	    }
	}





int main(int argc, char ** argv){

    char targetEncrypted[ENCRYPTED_LENGTH] = "kfiqnm1770";
    int found = 0;
    char resultPassword[PASSWORD_LENGTH + 1];

    // device memory.
    char *d_encryptedTarget, *d_resultPassword;
    int *d_found;

    hipMalloc(&d_encryptedTarget, ENCRYPTED_LENGTH);
    hipMalloc(&d_found, sizeof(int));
    hipMalloc(&d_resultPassword, PASSWORD_LENGTH + 1);

    hipMemcpy(d_encryptedTarget, targetEncrypted, ENCRYPTED_LENGTH, hipMemcpyHostToDevice);
    hipMemcpy(d_found, &found, sizeof(int), hipMemcpyHostToDevice);

    // Launching kernel
    kernel<<<dim3(1000,1,1), dim3(1000, 1, 1)>>>(d_encryptedTarget, d_resultPassword, d_found);

    // results back to host
    hipMemcpy(&found, d_found, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(resultPassword, d_resultPassword, PASSWORD_LENGTH + 1, hipMemcpyDeviceToHost);

    // Check result
    if (found) {
        printf("Password found: %s\n", resultPassword);
    } else {
        printf("Password not found.\n");
    }

    // Free device memory.
    hipFree(d_encryptedTarget);
    hipFree(d_found);
    hipFree(d_resultPassword);

    return 0;
}
