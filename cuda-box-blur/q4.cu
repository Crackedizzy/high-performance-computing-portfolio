#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "lodepng.h"

__global__ void Blur(unsigned char * inputImage, unsigned char * outputImage, int width, int height){
	
	int r=0, g=0, b=0, a=0, n_count=0;
	int threadID = blockDim.x * blockIdx.x + threadIdx.x;
	int x = threadID % width;
	int y = threadID / width;
	
	if (x >= width || y >= height) {
		return;
	}

	for (int i = -1; i <= 1; i++) {
		for (int j = -1;  j <= 1; j++) {
			int xi = x + i;
			int yj = y + j;
			
			if ( xi >= 0 && xi < width && yj >= 0 && yj < height){//making sure im not scanning outside of the image
				 
				int pixel = (yj * width + xi) * 4;
				r += inputImage[pixel];
				g += inputImage[pixel+1];
				b += inputImage[pixel+2];
				a += inputImage[pixel+3];
				n_count++;
			}
		}
	}
	
	int pixel = (y * width + x) * 4;//mult by width to know hw many pixels are b4 this row
	
		outputImage[pixel] = r/n_count;
		outputImage[pixel+1] = g/n_count;
		outputImage[pixel+2] = b/n_count;
		outputImage[pixel+3] = a;
	
}

int main(int argc,  char ** argv){
	
	unsigned char * cpuImage;
	unsigned int width, height;
	char * filename = argv[1];
	char * newFilename = argv[2];
	
	lodepng_decode32_file(&cpuImage, &width, &height, filename);
	
	int arraySize = width * height * 4;
	int memorySize = arraySize * sizeof(unsigned char);
	
	unsigned char * gpuInput;
	unsigned char * gpuOutput;
	
	unsigned char cpuOutput[arraySize];
	
	hipMalloc((void**) &gpuInput, memorySize);
	hipMalloc((void**) &gpuOutput, memorySize);
	
	hipMemcpy(gpuInput, cpuImage, memorySize, hipMemcpyHostToDevice); 
	
	Blur<<<24, 100>>>(gpuInput, gpuOutput, width, height);
	hipDeviceSynchronize();
	
	hipMemcpy(cpuOutput, gpuOutput, memorySize, hipMemcpyDeviceToHost); 
	
	lodepng_encode32_file(newFilename, cpuOutput, width, height);
	
	hipFree(gpuInput);
    	hipFree(gpuOutput);
	
	return 0;
}
